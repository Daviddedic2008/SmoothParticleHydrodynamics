#include "hip/hip_runtime.h"
﻿#include "particleKernel.cuh"
#include "stdinclude.cuh"
#include "sorts.cuh"
#include "dataTransfer.cuh"

#include "particleStructs.cuh"

inline __device__ float approximatePointDensity(const particle p) {
	const int xo = p.boxID % numCellsY;
	const int yo = p.boxID / numCellsY;
	float density = 0.0f;
	// once found, must divide by volume under smoothing function(to make smoothing radius irrelevant)

	#pragma unroll
	for (char i = 0, x = -1, y = -1; i < 9; i++, x++, y += (i % 3 == 0), x -= (x == 1) * 2) {
		if (xo + x >= 0 && xo + x < numCellsX && yo + y >= 0 && yo + y < numCellsY) {
			const int id = xo + x + (yo + y) * numCellsX;
			const int pid_start = (id > 0) ? frozenCountArr[id - 1] : 0;

			#pragma unroll
			for (int pi = pid_start; pi < frozenCountArr[id]; pi++) {
				const vec3 posDiff = p.pos - ((particle*)particles)[pi].pos;

				const float tmp = smoothingFunction(magnitude(posDiff));

				density += (tmp > 0) * tmp;
			}
		}
	}

	return density / lookupVolume; // volume under smoothing function
}

inline __device__ void applyForcesParticle(const int id) {
	if (id >= numParticles) { return; }
	particle& p = ((particle*)particles)[id];
	p.velocity.y += gravityConst;
	approximatePointDensity(p);
}

inline __device__ void addForceToPos(const int id) {
	particle p = ((particle*)particles)[id];
	const vec3 newPos = p.pos + p.velocity;

	const unsigned char inx = inXBounds(newPos);
	const unsigned char iny = inYBounds(newPos);
	const unsigned char inz = inZBounds(newPos);
	const unsigned char inv = inx & iny & inz;
	p.velocity.x = p.velocity.x * (-2 * !inx + 1) * (!inx * dampingFactor + inx);
	p.velocity.y = p.velocity.y * (-2 * !iny + 1) * (!iny * dampingFactor + iny);
	p.velocity.z = p.velocity.z * (-2 * !inz + 1) * (!inz * dampingFactor + inz);

	p.pos = newPos * inv + p.pos * !inv;

	((particle*)particles)[id] = p;
}

inline __device__ void recalculateBoundingBox(const int id) {
	const particle p = ((particle*)particles)[id];
	((particle*)particles)[id].recalcBox();
}

__global__ void updateParticleKernel() {
	const int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id >= numParticles) { return; }

	// add to velocity vector
	applyForcesParticle(id);

	// add velocity vector to pos vector
	addForceToPos(id);

	// move particle to new bounding box if needed
	recalculateBoundingBox(id);
}

void initBoundingVolumes() {
	initBoundingBoxes << <512, numCellsX* numCellsY* numCellsZ / 512 + 1 >> > ();
}

void updateLoop() {
	updateParticleKernel << <512, numParticles / 512 + 1 >> > ();
	//sortEvenOdd();
	radix();
	copyParticlesFromGPU();
	//quicksort(cpuParticleArr, 0 ,numParticles-1);
	//sendParticlesToGPU();
}