#include "hip/hip_runtime.h"
﻿#include "particleKernel.cuh"
#include "stdinclude.cuh"
#include "sorts.cuh"
#include "dataTransfer.cuh"

#include "particleStructs.cuh"

inline __device__ float approximatePointDensity(const particle p) {
	const int xo = p.boxID % numCellsY;
	const int yo = p.boxID / numCellsY;
	float density = 0.0f;
	// once found, must divide by volume under smoothing function(to make smoothing radius irrelevant)

	for (char i = 0, x = -1, y = -1; i < 9; i++, x++, y += (i % 3) == 0, x -= (x == 1) * 2) {
		if ((xo + x) >= 0 && (xo + x) < numCellsX && (yo + y) >= 0 && (yo + y) < numCellsY) {
			const int id = xo + x + (yo + y) * numCellsX;

			#pragma unroll
			for (int pi = (id > 0) ? frozenCountArr[id - 1] : 0; pi < frozenCountArr[id]; pi++) {
				const vec3 posDiff = p.pos - ((particle*)particles)[pi].pos;

				const float tmp = smoothingFunction(magnitude(posDiff));

				density += (tmp > 0) * tmp;
			}
		}
	}

	return density / lookupVolume; // volume under smoothing function
}

inline __global__ void applyPressureForce() {
	const int threadId = threadIdx.x + blockIdx.x * blockDim.x;

	particle& p = ((particle*)particles)[threadId];

	const int xo = p.boxID % numCellsY;
	const int yo = p.boxID / numCellsY;
	// once found, must divide by volume under smoothing function(to make smoothing radius irrelevant)

	

	for (char i = 0, x = -1, y = -1; i < 9; i++, x++, y += (i % 3) == 0, x -= (x == 1) * 2) {
		if ((xo + x) >= 0 && (xo + x) < numCellsX && (yo + y) >= 0 && (yo + y) < numCellsY) {
			const int id = xo + x + (yo + y) * numCellsX;

#pragma unroll

			for (int pi = (id > 0) ? frozenCountArr[id - 1] : 0; pi < frozenCountArr[id]; pi++) {
				particle& p2 = ((particle*)particles)[pi];

				

				//printf("%d\n", skp);

				vec3 distDiff = p2.pos - p.pos;


				const float dist = magnitude(distDiff);

				const bool skp = dist < 1e-5; // avoid branching

				distDiff = distDiff * (1 / dist);

				vec3 tmpForce = distDiff * ((p.density-targetDensity + p2.density-targetDensity) / (p2.density * 2e3) * smoothingFunctionDerivative(dist));

				tmpForce = skp ? vec3(0, 0, 0) : tmpForce;

				atomicAdd(&(p.velocity.x), tmpForce.x);
				atomicAdd(&(p.velocity.y), tmpForce.y);
				atomicAdd(&(p.velocity.z), tmpForce.z);

				atomicAdd(&(p2.velocity.x), -tmpForce.x);
				atomicAdd(&(p2.velocity.y), -tmpForce.y);
				atomicAdd(&(p2.velocity.z), -tmpForce.z);
			}
		}
	}
}

inline __device__ vec3 getDensityGradient(const particle p) {
	// returns scaled vector!
	// to compute gradient, it deosnt sample density directly with the function
	// it takes n random particles from the left, right, up, and down bounding boxes
	// it uses the averages of each box to save computation time and retain most accuracy
	// works best with smaller boxes
	// particles in boxes can be assumed to have random order(thread-dependent order)
	
}

inline __device__ void applyForcesParticle(const int id) {
	if (id >= numParticles) { return; }
	particle& p = ((particle*)particles)[id];
	p.velocity.y += gravityConst;
	p.density = approximatePointDensity(p);
}

inline __device__ void addForceToPos(const int id) {
	particle p = ((particle*)particles)[id];
	const vec3 newPos = p.pos + p.velocity;

	const unsigned char inx = inXBounds(newPos);
	const unsigned char iny = inYBounds(newPos);
	const unsigned char inz = inZBounds(newPos);
	const unsigned char inv = inx & iny & inz;
	p.velocity.x = p.velocity.x * (-2 * !inx + 1) * (!inx * dampingFactor + inx);
	p.velocity.y = p.velocity.y * (-2 * !iny + 1) * (!iny * dampingFactor + iny);
	p.velocity.z = p.velocity.z * (-2 * !inz + 1) * (!inz * dampingFactor + inz);

	p.pos = newPos * inv + p.pos * !inv;

	((particle*)particles)[id] = p;
}

inline __device__ void recalculateBoundingBox(const int id) {
	const particle p = ((particle*)particles)[id];
	((particle*)particles)[id].recalcBox();
}

__global__ void updateParticleKernel() {
	const int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id >= numParticles) { return; }

	// add to velocity vector
	applyForcesParticle(id);

	// add velocity vector to pos vector
	addForceToPos(id);

	// move particle to new bounding box if needed
	recalculateBoundingBox(id);
}

void initBoundingVolumes() {
	initBoundingBoxes << <512, numCellsX* numCellsY* numCellsZ / 512 + 1 >> > ();
}

void updateLoop() {
	updateParticleKernel << <512, numParticles / 512 + 1 >> > ();

	applyPressureForce << <512, numParticles / 512 + 1 >> > ();
	
	//sortEvenOdd();
	radix();

	copyParticlesFromGPU();
	//quicksort(cpuParticleArr, 0 ,numParticles-1);
	//sendParticlesToGPU();
}